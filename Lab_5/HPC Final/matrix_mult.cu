#include "hip/hip_runtime.h"
%%writefile matrix_mult.cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

void displayMatrix(int* mat, int rows, int cols, const char* name) {
    printf("\n%s:\n", name);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%4d ", mat[i * cols + j]);
        }
        printf("\n");
    }
}

//!nvcc -arch=sm_75 matrix_mult.cu -o matrix_mult
//!./matrix_mult
//


void matrix_mult(int* a, int* b, int* c, int rowsA, int colsA, int colsB) {
    for (int row = 0; row < rowsA; row++) {
        for (int col = 0; col < colsB; col++) {
            int sum = 0;
            for (int i = 0; i < colsA; i++) {
                sum += a[row * colsA + i] * b[i * colsB + col];
            }
            c[row * colsB + col] = sum;
        }
    }
}

__global__ void matrixMul(int* a, int* b, int* c, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < rowsA && col < colsB) {
        for (int i = 0; i < colsA; i++) {
            sum += a[row * colsA + i] * b[i * colsB + col];
        }
        c[row * colsB + col] = sum;
    }
}

bool verifyMatrixResults(int* c_cuda, int* c_normal, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        if (c_cuda[i] != c_normal[i]) {
            return false;
        }
    }
    return true;
}

int main() {
    int rowsA = 4;
    int colsA = 4;
    int rowsB = 4;
    int colsB = 4;

    int *a, *b, *c_cuda, *c_normal;
    int *dev_a, *dev_b, *dev_c;

    a = (int*)malloc(rowsA * colsA * sizeof(int));
    b = (int*)malloc(rowsB * colsB * sizeof(int));
    c_cuda = (int*)malloc(rowsA * colsB * sizeof(int));
    c_normal = (int*)malloc(rowsA * colsB * sizeof(int));

    for (int i = 0; i < rowsA * colsA; i++) {
        a[i] = rand() % 10;
    }
    for (int i = 0; i < rowsB * colsB; i++) {
        b[i] = rand() % 10;
    }

    hipMalloc((void**)&dev_a, rowsA * colsA * sizeof(int));
    hipMalloc((void**)&dev_b, rowsB * colsB * sizeof(int));
    hipMalloc((void**)&dev_c, rowsA * colsB * sizeof(int));

    hipMemcpy(dev_a, a, rowsA * colsA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, rowsB * colsB * sizeof(int), hipMemcpyHostToDevice);

    clock_t start_cuda = clock();

    dim3 blockSize(16, 16);
    dim3 gridSize((colsB + blockSize.x - 1) / blockSize.x, (rowsA + blockSize.y - 1) / blockSize.y);
    matrixMul<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, rowsA, colsA, colsB);
    hipMemcpy(c_cuda, dev_c, rowsA * colsB * sizeof(int), hipMemcpyDeviceToHost);

    clock_t end_cuda = clock();
    double cuda_time = (double)(end_cuda - start_cuda) / CLOCKS_PER_SEC;
    printf("Time Taken GPU : %f", cuda_time);

    clock_t start_normal = clock();
    matrix_mult(a, b, c_normal, rowsA, colsA, colsB);
    clock_t end_normal = clock();
    double normal_time = (double)(end_normal - start_normal) / CLOCKS_PER_SEC;
    printf("\nTime Taken CPU : %f ", normal_time);

    bool match = verifyMatrixResults(c_cuda, c_normal, rowsA, colsB);
    printf("\nOutput Match: %s", match ? "True" : "False");

    double speedup = normal_time / cuda_time;
    printf("\nSpeedup Factor: %f\n", speedup);

    // Display matrices (optional for small size)
    displayMatrix(a, rowsA, colsA, "Matrix A");
    displayMatrix(b, rowsB, colsB, "Matrix B");
    displayMatrix(c_cuda, rowsA, colsB, "Result from GPU");
    displayMatrix(c_normal, rowsA, colsB, "Result from CPU");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c_cuda);
    free(c_normal);

    return 0;
}


//!nvcc -arch=sm_75 matrix_mult.cu -o matrix_mult
//!./matrix_mult
